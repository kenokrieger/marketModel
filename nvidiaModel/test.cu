#include <iostream>
#include <string>

#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>

#include <hipcub/hipcub.hpp>
#define CUB_CHUNK_SIZE ((1ll<<31) - (1ll<<28))

#include "cudamacro.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""

#define THREADS 128

int main() {
		
		return 0;
}
