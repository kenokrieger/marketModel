#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 *
 */

#include <chrono>
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <hipblas.h>

#include <hipcub/hipcub.hpp>
#define CUB_CHUNK_SIZE ((1ll<<31) - (1ll<<28))

#include "cudamacro.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include ""

#define TCRIT 1.0f
#define THREADS  128

__device__ int GLOBAL_MARKET = 0;

__global__ void init_agents(signed char* agents,
                              const float* __restrict__ random_values,
                              const long long grid_height,
                              const long long grid_width) {
    // iterate over all agents in parallel and assign each of them
    // a strategy of either +1 or -1
    const long long  thread_id = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;

    // check for out of bound access
    if (thread_id >= grid_width * grid_height) return;

    // use random number between 0.0 and 1.0 generated beforehand
    float random = random_values[thread_id];
    agents[thread_id] = (random < 0.5f) ? -1 : 1;
}

template<bool is_black>
__global__ void update_agents(signed char* agents,
                              const signed char* __restrict__ checkerboard_agents,
                              const float* __restrict__ random_values,
                              const float alpha,
                              const float beta,
                              const float j,
                              const long long grid_height,
                              const long long grid_width) {
    const long long thread_id = static_cast<long long>(blockDim.x) * blockIdx.x + threadIdx.x;
    const int row = thread_id / grid_width;
    const int col = thread_id % grid_width;

    // check for out of bound access
    if (row >= grid_height || col >= grid_width) return;

    // determine nearest neighbors on the opposite grid
    int lower_neighbor_row = (row + 1 < grid_height) ? row + 1 : 0;
    int upper_neighbor_row = (row - 1 >= 0) ? row - 1: grid_height - 1;
    int right_neighbor_col = (col + 1 < grid_width) ? col + 1 : 0;
    int left_neighbor_col = (col - 1 >= 0) ? col - 1: grid_width - 1;

    // Select off-column index based on color and row index parity:
    // One of the neighbors will always have the exact same index
    // as the agents where as the remaining one will either have an
    // index differing by +1 or -1 depending on the position of the
    // agent on the grid
    int horizontal_neighbor_col;
    if (is_black) {
        horizontal_neighbor_col = (row % 2) ? right_neighbor_col : left_neighbor_col;
    } else {
        horizontal_neighbor_col = (row % 2) ? left_neighbor_col : right_neighbor_col;
    }
    // Compute sum of nearest neighbor spins:
    // Multiply the row with the grid-width to contain
    // the actual index in the array
    float neighbor_coupling = j * (
            checkerboard_agents[upper_neighbor_row * grid_width + col]
          + checkerboard_agents[lower_neighbor_row * grid_width + col]
          + checkerboard_agents[row * grid_width + col]
          + checkerboard_agents[row * grid_width + horizontal_neighbor_col]);

    signed char old_strategy = agents[row * grid_width + col];
    double market_coupling = -alpha / pow(grid_width, 2) * abs(GLOBAL_MARKET);
    double field = neighbor_coupling + market_coupling * old_strategy;
    // Determine whether to flip spin
    float probability = 1 / (1 + exp(-2.0 * beta * field));
    signed char new_strategy = random_values[row * grid_width + col] < probability ? -1 : 1;
    if (new_strategy != old_strategy)
        GLOBAL_MARKET -= 2 * old_strategy;
    agents[row * grid_width + col] = new_strategy;
}

// Write lattice configuration to file
void write_lattice(signed char *lattice_b, signed char *lattice_w, std::string filename, long long nx, long long ny) {
  printf("Writing lattice to %s...\n", filename.c_str());
  signed char *lattice_h, *lattice_b_h, *lattice_w_h;
  lattice_h = (signed char*) malloc(nx * ny * sizeof(*lattice_h));
  lattice_b_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_b_h));
  lattice_w_h = (signed char*) malloc(nx * ny/2 * sizeof(*lattice_w_h));

  CHECK_CUDA(hipMemcpy(lattice_b_h, lattice_b, nx * ny/2 * sizeof(*lattice_b), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(lattice_w_h, lattice_b, nx * ny/2 * sizeof(*lattice_w), hipMemcpyDeviceToHost));

  for (int i = 0; i < nx; i++) {
    for (int j = 0; j < ny/2; j++) {
      if (i % 2) {
        lattice_h[i*ny + 2*j+1] = lattice_b_h[i*ny/2 + j];
        lattice_h[i*ny + 2*j] = lattice_w_h[i*ny/2 + j];
      } else {
        lattice_h[i*ny + 2*j] = lattice_b_h[i*ny/2 + j];
        lattice_h[i*ny + 2*j+1] = lattice_w_h[i*ny/2 + j];
      }
    }
  }

  std::ofstream f;
  f.open(filename);
  if (f.is_open()) {
    for (int i = 0; i < nx; i++) {
      for (int j = 0; j < ny; j++) {
         f << (int)lattice_h[i * ny + j] << " ";
      }
      f << std::endl;
    }
  }
  f.close();

  free(lattice_h);
  free(lattice_b_h);
  free(lattice_w_h);
}

void update(signed char *lattice_b, signed char *lattice_w, float* randvals, hiprandGenerator_t rng, float alpha,
            float inv_temp, float j, long long nx, long long ny) {

  // Setup CUDA launch configuration
  int blocks = (nx * ny/2 + THREADS - 1) / THREADS;

  // Update black
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny/2));
  update_agents<true><<<blocks, THREADS>>>(lattice_b, lattice_w, randvals, alpha, inv_temp, j, nx, ny/2);

  // Update white
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny/2));
  update_agents<false><<<blocks, THREADS>>>(lattice_w, lattice_b, randvals, alpha, inv_temp, j, nx, ny/2);
}

int main() {
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;

    //Use command-line specified CUDA device, otherwise use device with highest Gflops/s
    // command-line input is given as 0, 0
    int dev = findCudaDevice(0, 0);

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    printf("CUDA device [%s] has %d Multi-Processors, Compute %d.%d\n",
        deviceProp.name, deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    // Defaults
    long long nx = 2048;
    long long ny = 2048;
    int nwarmup = 100;
    int niters = 1000;
    bool write = false;
    unsigned long long seed = 1234ULL;
    write = false;

  float inv_temp = 1 / 1.5f;

  // Setup cuRAND generator
  hiprandGenerator_t rng;
  CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));
  float *randvals;
  CHECK_CUDA(hipMalloc(&randvals, nx * ny/2 * sizeof(*randvals)));

  // Setup black and white lattice arrays on device
  signed char *lattice_b, *lattice_w;
  CHECK_CUDA(hipMalloc(&lattice_b, nx * ny/2 * sizeof(*lattice_b)));
  CHECK_CUDA(hipMalloc(&lattice_w, nx * ny/2 * sizeof(*lattice_w)));

  int blocks = (nx * ny/2 + THREADS - 1) / THREADS;
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny/2));
  init_agents<<<blocks, THREADS>>>(lattice_b, randvals, nx, ny/2);
  CHECK_CURAND(hiprandGenerateUniform(rng, randvals, nx*ny/2));
  init_agents<<<blocks, THREADS>>>(lattice_w, randvals, nx, ny/2);

  float alpha = 3.0f;
  float j = 1.0f;
  // Warmup iterations
  printf("Starting warmup...\n");
  for (int i = 0; i < nwarmup; i++) {
    update(lattice_b, lattice_w, randvals, rng, alpha, inv_temp, j, nx, ny);
  }

  CHECK_CUDA(hipDeviceSynchronize());

  printf("Starting trial iterations...\n");
  auto t0 = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < niters; i++) {
    update(lattice_b, lattice_w, randvals, rng, alpha, inv_temp, j, nx, ny);
    if (i % 1000 == 0) printf("Completed %d/%d iterations...\n", i+1, niters);
  }

  CHECK_CUDA(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();

  double duration = (double) std::chrono::duration_cast<std::chrono::microseconds>(t1-t0).count();
  printf("REPORT:\n");
  printf("\tnGPUs: %d\n", 1);
  printf("\ttemperature: %f * %f\n", alpha, TCRIT);
  printf("\tseed: %llu\n", seed);
  printf("\twarmup iterations: %d\n", nwarmup);
  printf("\ttrial iterations: %d\n", niters);
  printf("\tlattice dimensions: %lld x %lld\n", nx, ny);
  printf("\telapsed time: %f sec\n", duration * 1e-6);
  printf("\tupdates per ns: %f\n", (double) (nx * ny) * niters / duration * 1e-3);

  // Reduce
  double* devsum;
  int nchunks = (nx * ny/2 + CUB_CHUNK_SIZE - 1)/ CUB_CHUNK_SIZE;
  CHECK_CUDA(hipMalloc(&devsum, 2 * nchunks * sizeof(*devsum)));
  size_t cub_workspace_bytes = 0;
  void* workspace = NULL;
  CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, lattice_b, devsum, CUB_CHUNK_SIZE));
  CHECK_CUDA(hipMalloc(&workspace, cub_workspace_bytes));
  for (int i = 0; i < nchunks; i++) {
    CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, &lattice_b[i*CUB_CHUNK_SIZE], devsum + 2*i,
                           std::min((long long) CUB_CHUNK_SIZE, nx * ny/2 - i * CUB_CHUNK_SIZE)));
    CHECK_CUDA(hipcub::DeviceReduce::Sum(workspace, cub_workspace_bytes, &lattice_w[i*CUB_CHUNK_SIZE], devsum + 2*i + 1,
                           std::min((long long) CUB_CHUNK_SIZE, nx * ny/2 - i * CUB_CHUNK_SIZE)));
  }

  double* hostsum;
  hostsum = (double*)malloc(2 * nchunks * sizeof(*hostsum));
  CHECK_CUDA(hipMemcpy(hostsum, devsum, 2 * nchunks * sizeof(*devsum), hipMemcpyDeviceToHost));
  double fullsum = 0.0;
  for (int i = 0; i < 2 * nchunks; i++) {
    fullsum += hostsum[i];
  }
  std::cout << "\taverage magnetism (absolute): " << abs(fullsum / (nx * ny)) << std::endl;

  if (write) write_lattice(lattice_b, lattice_w, "final.txt", nx, ny);

  return 0;
}
